#include "hip/hip_runtime.h"
/* Simple code to check whether there a working CUDA runtime + driver + GPU device
 * combination present in the system.
 *
 * The expected result of this program is the CUDA runtime and driver API version
 * printed on the command line and a confirmation that a test kernel has been
 * successfully executed on the CUDA GPU.
 *
 * Compile with:    nvcc check-cuda-drv-runtime.cu -o chk
 * Then run:        ./chk
 * Expected outputs:
 * - everything working fine (CUDA 7.5 driver + runtime):
 *   CUDA driver version: 7050
 *   CUDA runtime version: 7050
 *   Test kernel executed successfully!
 *
 * - no device detected:
 *   CUDA driver version: 7050
 *   hipRuntimeGetVersion failed: no CUDA-capable device is detected
 *
 * - runtime / driver mismatch (driver ver < runtime ver):
 *   CUDA driver version: 7050
 *   hipRuntimeGetVersion failed: CUDA driver version is insufficient for CUDA runtime version
 *
 * Author: Szilárd Páll (sin.pecado@gmail.com)
 * 
 */

#include <cstdio>

__global__ void test_kernel() {}

static void check_cuda_retval(hipError_t status, const char* msg)
{
    if (status != hipSuccess)
    {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(status));
        exit(1);
    }
}

int main()
{
    hipError_t stat;
    int rt_ver = 0, drv_ver = 0;

    stat = hipDriverGetVersion(&drv_ver);
    check_cuda_retval(stat, "hipDriverGetVersion failed");
    printf("CUDA driver version: %d\n", drv_ver);

    stat = hipRuntimeGetVersion(&rt_ver);
    check_cuda_retval(stat, "hipRuntimeGetVersion failed");
    printf("CUDA runtime version: %d\n", rt_ver);

    test_kernel<<<1, 512, 0>>>();
    stat = hipDeviceSynchronize();
    check_cuda_retval(stat, "test kernel launch failed");
    printf("Test kernel executed successfully!\n");

    return 0;
}
